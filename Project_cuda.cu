////////////// ME766 PROJECT////////////////////////////////

////////////// Parallelization of PageRank Algorithm using CUDA and OpenMp  /////////////////////////

//// Instructor:
//// Shivasubramanian Gopalakrishnan 

//// Submitted by :
//// Arpit Tiwari
//// Ansh Thamke 
//// Raj Ingole 
//// Sumit Bhong


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include<stdlib.h>
#include<bits/stdc++.h>
#include <stdio.h>
#include<time.h>
#include <sys/time.h>
using namespace std;

#define TILE_WIDTH 32

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }						////////// Function to check error in device functions///////
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void matrixmult(float *a, float *b, float *c,int N,int d) 					/////////// Matrix Multiplication  a*v1=v2 ////////
{
	int row = blockIdx.y * blockDim.y +threadIdx.y;
	int col = blockIdx.x * blockDim.x +threadIdx.x;
	
	if(row < N && col < d)
	{	
		float temp=0;

		for(int p=0; p<N ; p++)
		{
			temp=temp + a[row * N + p] * b[p * d + col];

		}

		c[row*d + col] = temp ;
	}
	
}

float *length(float *a,float *len,int n)
{

for(int j=0;j<n;j++)
{
int sum=0;
	for(int i=0;i<n;i++)
	{
		if(a[i*n+j]>0){sum++;}
	}

len[j]=sum;
}
return len;
}

float *stochastic(float *a,float *len,float t,int n)		///////////// Formation of Stochastic Matrix/////////////
{
for(int i=0;i<n;i++)
{
float g=len[i];
	if(g>0){
				for(int j=0;j<n;j++)
				{
					a[j*n+i]=(a[j*n+i]*t)/(g);
				}
			}
	else {
for(int j=0;j<n;j++)
{ a[j*n+i]=(1.0*t)/n;  }
}

}
return a;
}

float *transmatrix(float *a,float t , int n)			////////////// Formation of Transformation matrix considering damping factor/////////
{
float b[n][n];
	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			b[i][j]=((1-t)*1.0)/n;
		}
	}
	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			a[i*n+j]=a[i*n+j]+b[i][j];
		}
	}
return a;
}

bool error(float *b, float *c,float tol,float sum,int n,int f,int d)	////////// Function to check Convergence /////////
{
	for(int i=0;i<n;i++)
	{
		sum =sum + (c[d*i+f]-b[d*i+f])*(c[d*i+f]-b[d*i+f]);
	}
if(sum<tol)
{
return false;
}
else return true;
}

int main(){														////// Main function starts //////
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
  
  
  int n=100;                        /////// NUMBER OF NODES	///////////
  int d=15;
  
  float tol=0.00000000005;           //////////// TOLERANCE VALUE	////////
  float *len;
  
  float b=0.85;                       ////////// Damping Factor	//////////
  float *v1,*v2,*matri;
  
  size_t bytes = n*n*sizeof(float);
  
	  matri = (float*)malloc( bytes );                       ////// DYNAMIC MEMORY ALLOCATION /////////
		v1 = (float*)malloc( n*d*sizeof(float) );
	  v2 = (float*)malloc(  n*d*sizeof(float) );
		len = (float*)malloc( n*sizeof(float));
	
	gpuErrchk( hipMallocManaged(&matri,bytes));				//////// Device Memory Allocation///////
	gpuErrchk(hipMallocManaged(&v1,d*n*sizeof(float)));
	gpuErrchk(hipMallocManaged(&v2,d*n*sizeof(float)));
	

for(int i=0;i<d*n;i++){
v1[i]=1.0/n;
v2[i]=1.0;}


///////
int i=0;
float *first,*second;
int t=291;
first = (float*)malloc( t*sizeof(float));
second = (float*)malloc( t*sizeof(float));
string line;
ifstream myfile("Barbasi.txt");
while(std::getline(myfile,line))
{
std::stringstream linestream(line);

float val1;
float val2;

while(linestream>>val1>>val2){

first[i]=val1;
second[i]=val2;
}
i++;
}
myfile.close();




int f1,f2;
for(int i=0;i<n;i++)
{
    for(int j=0;j<n;j++)
    {
        matri[i*n+j]=0;
     }
 }


for(int i=0;i<t;i++){


f1=first[i];
f2=second[i];
matri[f2*n+f1]=1;
}

length(matri,len,n);
stochastic(matri,len,b,n);
transmatrix(matri,b,n);

int blocks = (n+TILE_WIDTH-1)/TILE_WIDTH;				////////// Number of blocks to be used///////
	
	dim3 dim_block(TILE_WIDTH, TILE_WIDTH);				////////// Number of threads per block ///////
	dim3 dim_grid( blocks, blocks);

matrixmult<<<dim_grid, dim_block>>>(matri, v1, v2, n,d);		///////// Cuda function call /////
hipDeviceSynchronize();

int f=0;

  
while(error(v1,v2,tol,0,n,f,d))									/////// Iterative Multiplication /////
{
    for(int i=0;i<n;i++){v1[i*d+f+1]=v2[i*d+f];}

	matrixmult<<<dim_grid, dim_block>>>(matri, v1, v2, n,d);
	hipDeviceSynchronize();
	
f++;
}
double coutnnn=0;										
for(int i=0;i<n;i++){
//cout<<v2[i*d+f]<<endl;
coutnnn=coutnnn+v2[i*d+f];
}

cout<<coutnnn<<endl;
cout<<"Number of iterations to converge = "<<f<<endl;

cout<<endl<<endl;
gettimeofday(&t2, 0);

double timee = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;		////////// Time taken to to calculate final PageRank//////
cout<<timee/1000<<endl;

}
